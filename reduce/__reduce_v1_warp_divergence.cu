#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256
const float eps = 1e-3;

__global__ void reduce1(float* in, float* out) {
    extern __shared__ float sdata[];
    const int tid = threadIdx.x;
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    sdata[tid] = in[idx];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x)
            sdata[index] += sdata[index + s];
        __syncthreads();
    }
    if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}

bool check(float *out,float *res,int n){
    for(int i = 0; i < n; i++){
        if(abs(out[i] - res[i]) > eps)
            return false;
    }
    return true;
}

int main() {
    const int N = 32 * 1024 * 1024;
    float *a= (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK;
    float *out=(float *)malloc((N / THREAD_PER_BLOCK) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out,(N / THREAD_PER_BLOCK) * sizeof(float));
    float *res = (float *)malloc((N / THREAD_PER_BLOCK) * sizeof(float));
    
    for (int i = 0; i < N; i++)
        a[i] = 2.0 * (float)drand48() - 1.0;
    
    // calculate on cpu
    for (int i = 0; i < block_num; ++i) {
        float cur = 0.0f;
        for (int j = 0; j < THREAD_PER_BLOCK; ++j){
            cur += a[i * THREAD_PER_BLOCK + j];
        }
        res[i] = cur;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(N / THREAD_PER_BLOCK);
    dim3 Block(THREAD_PER_BLOCK);
    int smem = THREAD_PER_BLOCK * sizeof(float);

    reduce1<<<Grid, Block, smem>>>(d_a, d_out);

    hipMemcpy(out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if(check(out, res, block_num)) printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i = 0;i < block_num; i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);
    return 0;
}
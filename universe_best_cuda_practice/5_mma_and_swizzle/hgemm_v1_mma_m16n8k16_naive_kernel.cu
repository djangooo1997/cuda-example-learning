#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>

#include "common/tester.h"
#include "common/common.h"

using namespace nvcuda;
#define WARP_SIZE 32
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2 *>(&(value))[0])

#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1) asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" : "=r"(RD0), "=r"(RD1) : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

// only 1 warp per block(32 threads), m16n8k16. A, B, C: all row_major.
template <const int MMA_M = 16, const int MMA_N = 8, const int MMA_K = 16>
__global__ void hgemm_mma_m16n8k16_naive_kernel(half *A, half *B, half *C,
                                                int M, int N, int K)
{
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int NUM_K_TILES = div_ceil(K, MMA_K);
    constexpr int BM = MMA_M; // 16
    constexpr int BN = MMA_N; // 8
    constexpr int BK = MMA_K; // 16

    __shared__ half s_a[MMA_M][MMA_K]; // 16x16
    __shared__ half s_b[MMA_K][MMA_N]; // 16x8
    __shared__ half s_c[MMA_M][MMA_N]; // 16x8

    const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
    const int lane_id = tid % WARP_SIZE;                    // 0~31

    // s_a[16][16], 每行16，每线程load 8，需要2线程，共16行，需2x16=32线程
    const int load_smem_a_m = tid / 2;       // row 0~15
    const int load_smem_a_k = (tid % 2) * 8; // col 0,8
    // s_b[16][8], 每行8，每线程load 8，需要1线程，共16行，需16线程，只需一半线程加载
    const int load_smem_b_k = tid;                     // row 0~31, but only use 0~15
    const int load_smem_b_n = 0;                       // col 0
    const int load_gmem_a_m = by * BM + load_smem_a_m; // global m
    const int load_gmem_b_n = bx * BN + load_smem_b_n; // global n
    if (load_gmem_a_m >= M && load_gmem_b_n >= N)
        return;

    uint32_t RC[2] = {0, 0};

#pragma unroll
    for (int k = 0; k < NUM_K_TILES; ++k)
    {
        // gmem_a -> smem_a
        int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
        int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
        LDST128BITS(s_a[load_smem_a_m][load_smem_a_k]) = (LDST128BITS(A[load_gmem_a_addr]));

        // gmem_b -> smem_b
        if (lane_id < MMA_K)
        {
            int load_gmem_b_k = k * MMA_K + load_smem_b_k; // global row of b
            int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
            LDST128BITS(s_b[load_smem_b_k][load_smem_b_n]) = (LDST128BITS(B[load_gmem_b_addr]));
        }
        __syncthreads();

        uint32_t RA[4];
        uint32_t RB[2];

        // ldmatrix for s_a, ldmatrix.trans for s_b.
        // s_a: (0,1)*8 -> 0,8 -> [(0~15),(0,8)]
        uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
            &s_a[lane_id % 16][(lane_id / 16) * 8]);
        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], load_smem_a_ptr);
        uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
            &s_b[lane_id % 16][0]);
        LDMATRIX_X2_T(RB[0], RB[1], load_smem_b_ptr);

        HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);

        __syncthreads();
    }

    // s_c[16][8], https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
    // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
    // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
    LDST32BITS(s_c[lane_id / 4][(lane_id % 4) * 2]) = LDST32BITS(RC[0]);
    LDST32BITS(s_c[lane_id / 4 + 8][(lane_id % 4) * 2]) = LDST32BITS(RC[1]);

    __syncthreads();

    // store s_c[16][8]
    if (lane_id < MMA_M)
    {
        // store 128 bits per memory issue.
        int store_gmem_c_m = by * BM + lane_id;
        int store_gmem_c_n = bx * BN;
        int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
        LDST128BITS(C[store_gmem_c_addr]) = (LDST128BITS(s_c[lane_id][0]));
    }
}
void hgemm_mma_m16n8k16_naive(half *A, half *B, half *C, int M, int N, int K)
{
    constexpr int WMMA_M = 16;
    constexpr int WMMA_N = 8;
    constexpr int WMMA_K = 16;
    dim3 block(32);
    dim3 grid(div_ceil(N, WMMA_N), div_ceil(M, WMMA_M));

    hgemm_mma_m16n8k16_naive_kernel<WMMA_M, WMMA_N, WMMA_K><<<grid, block>>>(A, B, C, M, N, K);
}

int main(int argc, char *argv[])
{
    Tester tester(512, 2048, 1024, 1, 10, 100, false);
    tester.evaluate(hgemm_mma_m16n8k16_naive, "hgemm_mma_m16n8k16_naive");
    return 0;
}
#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>

#include "common/tester.h"
#include "common/common.h"

#define WARP_SIZE 32
#define LDST32BITS(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

using namespace nvcuda;

// Double buffers
template <const int WMMA_M = 16, const int WMMA_N = 16, const int WMMA_K = 16,
          const int WMMA_TILE_M = 4, const int WMMA_TILE_N = 2,
          const int WARP_TILE_M = 2, const int WARP_TILE_N = 4,
          const int OFFSET = 0>
__global__ void hgemm_wmma_m16n16k16_mma4x2_warp2x4_dbuf_async_kernel(
    half *A, half *B, half *C, int M, int N, int K)
{
    // 256 threads(8 warps) per block.
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int NUM_K_TILES = div_ceil(K, WMMA_K);
    constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4*2=128
    constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2*4=128
    constexpr int BK = WMMA_K;                             // 16
    // 16x128x2=4KB, 4+4=8KB, padding to reduce bank conflicts.
    __shared__ half s_a[2][BM][BK + OFFSET], s_b[2][BK][BN + OFFSET];

    // 要保证相同的warp下thread执行相同的指令
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
    const int lane_id = tid % WARP_SIZE; // 0~31
    const int warp_m = warp_id / 2;      // 0,1,2,3
    const int warp_n = warp_id % 2;      // 0,1

    // 0. 先计算shared memory中的索引
    // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
    // 对于s_a每行16个数据，每个线程读取8个，需要2个线程；总共128行，需要128x2刚好256线程
    int load_smem_a_m = tid / 2;                // row 0~127
    int load_smem_a_k = (tid % 2 == 0) ? 0 : 8; // col 0,8
    // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=16 BN=128 按行读取 B行主序
    // 对于s_b每行128个数据，每个线程读8个数据，需要16个线程；总共16行，需要16x16=256个线程
    int load_smem_b_k = tid / 16;       // row 0~15
    int load_smem_b_n = (tid % 16) * 8; // col 0,8,...,120
    // 1. 再计算全局内存中的索引
    // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
    int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
    int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
    if (load_gmem_a_m >= M || load_gmem_b_n >= N)
        return;

    wmma::fragment<wmma::accumulator,
                   WMMA_M, WMMA_N, WMMA_K,
                   half>
        C_frag[WARP_TILE_M][WARP_TILE_N];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        A_frag[WARP_TILE_M];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        B_frag[WARP_TILE_N];

#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i)
    {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j)
        {
            wmma::fill_fragment(C_frag[i][j], 0.0);
        }
    }

    // k = 0 is loading here, buffer 0
    {
        int load_gmem_a_k = load_smem_a_k; // global col of a
        int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
        int load_gmem_b_k = load_smem_b_k; // global row of b
        int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

        uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
            &s_a[0][load_smem_a_m][load_smem_a_k]);
        CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

        uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
            &s_b[0][load_smem_b_k][load_smem_b_n]);
        CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
        CP_ASYNC_COMMIT_GROUP();
        CP_ASYNC_WAIT_GROUP(0);
    }
    __syncthreads();

#pragma unroll
    for (int k = 1; k < NUM_K_TILES; ++k)
    {                               // start from 1
        int smem_sel = (k - 1) & 1; // k 1->0, k 2->1, k 3->0, ...
        int smem_sel_next = k & 1;  // k 1->1, k 2->0, k 3->1, ...

        int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
        int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
        int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
        int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

        uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
            &s_a[smem_sel_next][load_smem_a_m][load_smem_a_k]);
        CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

        uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
            &s_b[smem_sel_next][load_smem_b_k][load_smem_b_n]);
        CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
        CP_ASYNC_COMMIT_GROUP(); //!!!!
#pragma unroll
        for (int i = 0; i < WARP_TILE_M; ++i)
        {
            // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
            const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
            wmma::load_matrix_sync(A_frag[i], &s_a[smem_sel][warp_smem_a_m][0], BK + OFFSET);
        }

#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j)
        {
            // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
            const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
            wmma::load_matrix_sync(B_frag[j], &s_b[smem_sel][0][warp_smem_b_n], BN + OFFSET);
        }

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; ++i)
        {
#pragma unroll
            for (int j = 0; j < WARP_TILE_N; ++j)
            {
                wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
            }
        }

        CP_ASYNC_WAIT_GROUP(0);

        __syncthreads();
    }

    // processing last k tile
    {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                       wmma::row_major>
            A_frag[WARP_TILE_M];
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                       wmma::row_major>
            B_frag[WARP_TILE_N];

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; ++i)
        {
            // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
            const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
            wmma::load_matrix_sync(A_frag[i], &s_a[1][warp_smem_a_m][0], BK + OFFSET);
        }

#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j)
        {
            // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
            const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
            wmma::load_matrix_sync(B_frag[j], &s_b[1][0][warp_smem_b_n], BN + OFFSET);
        }

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; ++i)
        {
#pragma unroll
            for (int j = 0; j < WARP_TILE_N; ++j)
            {
                wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
            }
        }
    }

// finally, store back to C matrix.
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i)
    {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j)
        {
            const int store_gmem_a_m = by * BM + warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
            const int store_gmem_a_n = bx * BN + warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
            wmma::store_matrix_sync(C + store_gmem_a_m * N + store_gmem_a_n, C_frag[i][j], N,
                                    wmma::mem_row_major);
        }
    }
}

void hgemm_wmma_m16n16k16_mma4x2_warp2x4_dbuf_async(half *A, half *B, half *C, int M, int N, int K)
{
    constexpr int WMMA_M = 16;
    constexpr int WMMA_N = 16;
    constexpr int WMMA_K = 16;
    constexpr int WMMA_TILE_M = 4;
    constexpr int WMMA_TILE_N = 2;

    constexpr int WARP_TILE_M = 2;
    constexpr int WARP_TILE_N = 4;

    dim3 block(256);
    dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));

    hgemm_wmma_m16n16k16_mma4x2_warp2x4_dbuf_async_kernel<WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M, WARP_TILE_N, 8><<<grid, block>>>(A, B, C, M, N, K);
}

int main(int argc, char *argv[])
{
    Tester tester(512, 2048, 1024, 1, 10, 100, true);
    tester.evaluate(hgemm_wmma_m16n16k16_mma4x2_warp2x4_dbuf_async, "hgemm_wmma_m16n16k16_mma4x2_warp2x4_dbuf_async");
    return 0;
}
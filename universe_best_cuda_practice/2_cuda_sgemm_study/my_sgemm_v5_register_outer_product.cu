
#include <hip/hip_runtime.h>
#include <cstdio>
#define A(i, j) a[(i) * n + (j)]
#define B(i, j) b[(i) * n + (j)]

void random_matrix(int m, int n, float *a)
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
#if 1
            A(i, j) = 2.0 * (float)drand48() - 1.0;
#else
            A(i, j) = (j - i) % 3;
#endif
}

float compare_matrices(int m, int n, float *a, float *b)
{
    int i, j;
    float max_diff = 0.0, diff;
    int printed = 0;

    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            diff = abs(A(i, j) - B(i, j));
            max_diff = (diff > max_diff ? diff : max_diff);
            if (0 == printed)
                if (max_diff > 0.5f || max_diff < -0.5f)
                {
                    printf("\n error: i %d  j %d diff %f  got %f  expect %f ", i, j, max_diff, A(i, j), B(i, j));
                    printed = 1;
                }
        }
    }
    return max_diff;
}

void cpu_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            float temp = 0.f;
            for (int k = 0; k < K; k++)
            {
                temp += A_ptr[m * K + k] * B_ptr[k * N + n];
            }
            C_ptr[m * N + n] = temp;
        }
    }
}

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

template <unsigned int M_NUM_PER_BLOCK,
          unsigned int N_NUM_PER_BLOCK,
          unsigned int K_NUM_PER_BLOCK,
          unsigned int NUM_PER_THREAD>
__global__ void cuda_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
    int ctx = tid % 16;
    int cty = tid / 16;
    float *A_ptr_start = A_ptr + blockIdx.y * M_NUM_PER_BLOCK * K;
    float *B_ptr_start = B_ptr + blockIdx.x * N_NUM_PER_BLOCK;

    __shared__ float a_shared[M_NUM_PER_BLOCK][K_NUM_PER_BLOCK];
    __shared__ float b_shared[K_NUM_PER_BLOCK][N_NUM_PER_BLOCK];

    constexpr int REG_NUM = NUM_PER_THREAD / 2;
    float a_reg[REG_NUM] = {0.f};
    float b_reg[REG_NUM] = {0.f};
    float temp[REG_NUM][REG_NUM] = {0.f};

    for (int s = 0; s < K; s += K_NUM_PER_BLOCK)
    {
        FETCH_FLOAT4(a_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOAT4(A_ptr_start[K * ty + s + tx * NUM_PER_THREAD]);
        FETCH_FLOAT4(b_shared[ty][tx * NUM_PER_THREAD]) = FETCH_FLOAT4(B_ptr_start[N * (ty + s) + tx * NUM_PER_THREAD]);
        __syncthreads();

        for (int k = 0; k < K_NUM_PER_BLOCK; k++)
        {
            a_reg[0] = a_shared[cty * 2][k];
            a_reg[1] = a_shared[cty * 2 + 1][k];
            b_reg[0] = b_shared[k][ctx * 2];
            b_reg[1] = b_shared[k][ctx * 2 + 1];
            for (int i = 0; i < REG_NUM; i++)
                for (int j = 0; j < REG_NUM; j++)
                    temp[i][j] += a_reg[i] * b_reg[j];
        }
        __syncthreads();
    }

    float *C_ptr_start = C_ptr + N * blockIdx.y * M_NUM_PER_BLOCK +
                         blockIdx.x * N_NUM_PER_BLOCK;
    for (int i = 0; i < REG_NUM; i++)
        for (int j = 0; j < REG_NUM; j++)
            C_ptr_start[N * (cty * 2 + i) + ctx * 2 + j] = temp[i][j];
}

int main()
{
    int m = 1024;
    int n = 1024;
    int k = 1024;
    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = k * n * sizeof(float);
    const size_t mem_size_C = m * n * sizeof(float);

    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    random_matrix(m, k, matrix_A_host);
    random_matrix(k, n, matrix_B_host);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C);

    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);

    constexpr int M_NUM_PER_BLOCK = 32;
    constexpr int N_NUM_PER_BLOCK = 32;
    constexpr int K_NUM_PER_BLOCK = 32;
    constexpr int NUM_PER_THREAD = 4;

    dim3 block(8, 32);
    dim3 grid(m / M_NUM_PER_BLOCK, n / N_NUM_PER_BLOCK);

    cuda_sgemm<M_NUM_PER_BLOCK, N_NUM_PER_BLOCK, K_NUM_PER_BLOCK, NUM_PER_THREAD><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);

    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);

    float diff = compare_matrices(m, n, matrix_C_host_gpu_calc, matrix_C_host_cpu_calc);
    if (diff > 0.5f || diff < -0.5f)
    {
        printf("diff too big !\n");
        exit(-1);
    }
    else
    {
        printf("right\n");
    }

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
    return 0;
}
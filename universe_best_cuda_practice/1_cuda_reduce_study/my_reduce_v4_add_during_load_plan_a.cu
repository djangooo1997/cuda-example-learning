#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

__global__ void reduce(float *d_input, float *d_output)
{
    __shared__ float shared[THREAD_PER_BLOCK];
    float *input_begin = d_input + blockDim.x * blockIdx.x * 2;
    shared[threadIdx.x] = input_begin[threadIdx.x] + input_begin[threadIdx.x + blockDim.x];
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i /= 2)
    {
        if (threadIdx.x < i)
            shared[threadIdx.x] += shared[threadIdx.x + i];
        __syncthreads();
    }
    // if (threadIdx.x == 0 or 2 or 4 or 6)
    //     input_begin[threadIdx.x] += input_begin[threadIdx.x + 1];
    // if (threadIdx.x == 0 or 4)
    //     input_begin[threadIdx.x] += input_begin[threadIdx.x + 2];
    // if (threadIdx.x == 0)
    //     input_begin[threadIdx.x] += input_begin[threadIdx.x + 4];
    if (threadIdx.x == 0)
        d_output[blockIdx.x] = shared[0];
}

bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 0.005)
            return false;
    }
    return true;
}

int main()
{
    // printf("hello reduce\n");
    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK / 2;
    float *output = (float *)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));
    float *result = (float *)malloc(block_num * sizeof(float));
    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }
    // cpu calc
    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < 2 * THREAD_PER_BLOCK; j++)
        {
            cur += input[i * 2 * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);
    for (int i = 0; i < 10; i++)
        reduce<<<Grid, Block>>>(d_input, d_output);
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(output, result, block_num))
        printf("the ans is right\n");
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
// "command" :
// "/usr/local/cuda-12.2/bin/nvcc
// -forward-unknown-to-host-compiler
// -isystem=/usr/local/cuda-12.2/include
// -g
// --generate-code=arch=compute_52,code=[compute_52,sm_52]
// -G
// -x cu
// -dc /home/hongkailin/universe_best_cuda_practice/1_cuda_reduce_study/my_reduce_v0_global_memory.cu
// -o CMakeFiles/my_reduce_v0_global_memory.dir/my_reduce_v0_global_memory.cu.o",